#include "hip/hip_runtime.h"
/* Flood fill
 *
 * The basic idea is very simple; each iteration we examine each pixel;
 * it the pixel borders a filled pixel, we test it, and if the test
 * succeeds, then we fill it, also setting a global flag that something
 * has changed.  We repeat this until nothing changes.
 *
 * That implementation turned out to be very slow... One problem is
 * that unfilled pixels have to examine all of their neighbors.
 * We will try a second implementation in which when a pixel is
 * filled, it marks it's un-filled neighbors.
 *
 * No difference.  Eliminating the flag checks after each kernel
 * launch reduces the time (for 100 iterations) from 11 msec to 7 msec!
 * This could probably be speeded up quite a bit if the control
 * logic could be run on the device instead of on the host...
 *
 * But can we launch a thread array from a device function?
 * Or should we launch the whole grid and have one special thread
 * which is the master?
 * The slow implementation has one thread per pixel in the image;
 * but many iterations are required... better perhaps to have one
 * thread per filled pixel with unchecked neighbors?
 *
 * We can only synchronize threads within a block, so we would have to
 * do this with a single block.  Let's say we have one thread per
 * filled pixel...  Each pixel has up to 4 fillable neighbors (although
 * only the first seed pixel with have all 4 unfilled).  So we have
 * an array in shared memory that we fill with the pixel values. (Need
 * to check how to avoid bank conflicts!)  Then we have a table of
 * of future pixels.  Each thread gets 4 slots.  After these have
 * been filled, we would like to prune duplicates; we won't have many
 * when filling parallel to a coordinate axis, but there will be lots
 * for an oblique front...  we could use a hash function?  Or use the
 * flag image.  We could use these values:
 * 0 - unchecked
 * 1 - filled
 * 2 - queued
 * 3 - rejected
 *
 *	0 0 0 0 0    0 0 0 0 0    0 0 2 0 0
 *	0 0 0 0 0    0 0 2 0 0    0 2 1 2 0
 *	0 0 2 0 0 -> 0 2 1 2 0 -> 2 1 1 1 2
 *	0 0 0 0 0    0 0 2 0 0    0 2 1 2 0
 *	0 0 0 0 0    0 0 0 0 0    0 0 2 0 0
 *
 * Shared memory per block is only 16k, so we can't put the whole image
 * there...
 *
 * We have an array of pixels to check, sized 4 times the max number
 * of threads in a block.  We have an array of active pixels, sized
 * the max number of threads.  After syncing the threads, we need to make
 * up the new active pixel list.  We may not have enough threads to do all
 * of the pixels, so we have several lists.  After processing each list,
 * we transfer new pixels to be checked to the list, marking them as queued.
 * If we run out of space, we will have to set a flag that says we
 * have unrecorded pixels that need to be queued; if that is set when
 * we are all done, we should scan the entire image again looking for them,
 * maybe using a special flag value to indicated un-fulfilled queue request?
 * If we can allocate 2048 queue request slots it ought to be enough
 * for a 512x512 image...
 *
 * We probably want to have the shared memory allocated at launch time...
 */

#include "quip_config.h"

#ifdef HAVE_CUDA

#define BUILD_FOR_CUDA

#include <stdio.h>
#include <hiprand.h>

#include "quip_prot.h"
#include "my_cu2.h"
#include "cuda_supp.h"			// describe_cuda_error

#include "cu2_fill_expanded.c"

#ifdef FOOBAR
#include "veclib/gpu_call_utils.h"
#include "veclib/slow_defs.h"
#include "../cu2/cu2_host_call_defs.h"
#include "../cu2/cu2_kern_call_defs.h"
//#include "my_vector_functions.h"	// max_threads_per_block

#define CHECK_CUDA_RETURN_VAL(msg)		\
	if( e != hipSuccess ){			\
		NWARN(msg);			\
	}

// The fill routine kernel

#define FILL_IF					\
	if( fabs( dst - v ) < tol ){		\
		src1 = 1;			\
		dst = fill_val;			\
		*flag = 1;			\
		return;				\
	}

// why not use vset ???

KERNEL_FUNC_QUALIFIER void zeroit(dim5 szarr, unsigned char* a, dim5 len )
{
	//int x,y;
	DECL_INDICES_1

	//x = blockIdx.x * blockDim.x + threadIdx.x;
	//y = blockIdx.y * blockDim.y + threadIdx.y;
	SET_INDICES_1

	/*
	if( x < len.x && y < len.y ){
		a[x+y*len.y] = 0;
	}
	*/
	dst = 0;
}

KERNEL_FUNC_QUALIFIER void g_sp_ifl_incs(dim5 szarr, float* a, dim5 inc1,
			unsigned char* b, dim5 inc2,
			dim5 len,
			float v, float tol, float fill_val, int *flag)
{
	/* BLOCK_VARS_DECLS */
	//INIT_INDICES_XYZ_2
	DECL_INDICES_2

	SET_INDICES_2

	//if (index1.x < len.x && index1.y < len.y ) {
		//SCALE_INDICES_XYZ_2
		if( src1 == 0 ){	// not filled yet
			// check each neighbor if filled
			if( index2.d5_dim[1] > 0 ){	// in-bounds
				index2.d5_dim[1] -= inc2.d5_dim[1];
				if( src1 > 0 ){	// neighbor filled?
					index2.d5_dim[1] += inc2.d5_dim[1];
					FILL_IF
				}
				index2.d5_dim[1] += inc2.d5_dim[1];
			}
			if( index2.d5_dim[1] < (len.d5_dim[1]-1)*inc2.d5_dim[1] ){
				index2.d5_dim[1] += inc2.d5_dim[1];
				if( src1 > 0 ){
					index2.d5_dim[1] -= inc2.d5_dim[1];
					FILL_IF
				}
				index2.d5_dim[1] -= inc2.d5_dim[1];
			}
			if( index2.d5_dim[2] > 0 ){
				index2.d5_dim[2] -= inc2.d5_dim[2];
				if( src1 > 0 ){
					index2.d5_dim[2] += inc2.d5_dim[2];
					FILL_IF
				}
				index2.d5_dim[2] += inc2.d5_dim[2];
			}
			if( index2.d5_dim[2] < (len.d5_dim[2]-1)*inc2.d5_dim[2] ){
				index2.d5_dim[2] += inc2.d5_dim[2];
				if( src1 > 0 ){
					index2.d5_dim[2] -= inc2.d5_dim[2];
					FILL_IF
				}
				index2.d5_dim[2] -= inc2.d5_dim[2];
			}
		}
	//}
}

__constant__ float test_value[1];
__constant__ float tolerance[1];
__constant__ float fill_value[1];

#define FILL_IF2					\
	if( fabs( dst - test_value[0] ) < tolerance[0] ){	\
		src1 = 1;				\
		dst = fill_value[0];			\
		return;					\
	}

KERNEL_FUNC_QUALIFIER void g_sp_ifl2_incs(dim5 szarr, float* a, dim5 inc1,
			unsigned char* b, dim5 inc2,
			dim5 len)
{
	/* BLOCK_VARS_DECLS */
	//INIT_INDICES_XYZ_2
	DECL_INDICES_2

	SET_INDICES_2

	if (index1.d5_dim[1] < len.d5_dim[1] && index1.d5_dim[2] < len.d5_dim[2] ) {
		//SCALE_INDICES_XYZ_2
		if( src1 == 0 ){	// not filled yet
			// check each neighbor if filled
			if( index2.d5_dim[1] > 0 ){	// in-bounds
				index2.d5_dim[1] -= inc2.d5_dim[1];
				if( src1 > 0 ){	// neighbor filled?
					index2.d5_dim[1] += inc2.d5_dim[1];
					FILL_IF2
				}
				index2.d5_dim[1] += inc2.d5_dim[1];
			}
			if( index2.d5_dim[1] < (len.d5_dim[1]-1)*inc2.d5_dim[1] ){
				index2.d5_dim[1] += inc2.d5_dim[1];
				if( src1 > 0 ){
					index2.d5_dim[1] -= inc2.d5_dim[1];
					FILL_IF2
				}
				index2.d5_dim[1] -= inc2.d5_dim[1];
			}
			if( index2.d5_dim[2] > 0 ){
				index2.d5_dim[2] -= inc2.d5_dim[2];
				if( src1 > 0 ){
					index2.d5_dim[2] += inc2.d5_dim[2];
					FILL_IF2
				}
				index2.d5_dim[2] += inc2.d5_dim[2];
			}
			if( index2.d5_dim[2] < (len.d5_dim[2]-1)*inc2.d5_dim[2] ){
				index2.d5_dim[2] += inc2.d5_dim[2];
				if( src1 > 0 ){
					index2.d5_dim[2] -= inc2.d5_dim[2];
					FILL_IF2
				}
				index2.d5_dim[2] -= inc2.d5_dim[2];
			}
		}
	}
}

void h_sp_ifl( Data_Obj *dp, int x, int y, float tol, float fill_val )
{
	BLOCK_VARS_DECLS
	hipError_t e;
	dim5 len, inc1, inc2;
	unsigned char *filled, b_one;
	float *f_p, v;
	int h_flag, *flag_p;
	int n_iterations;
	Vector_Args va1, *vap=(&va1);
	dim5 szarr;

	len.d5_dim[1] = OBJ_COLS(dp);
	len.d5_dim[2] = OBJ_ROWS(dp);

	//GET_MAX_THREADS(dp)
	SETUP_BLOCKS_XYZ(OBJ_PFDEV(dp))

	inc1.d5_dim[1] = OBJ_TYPE_INC(dp,1);
	inc1.d5_dim[2] = OBJ_TYPE_INC(dp,2);
	inc1.d5_dim[0] = inc1.d5_dim[3] = inc1.d5_dim[4] = 0;
	inc2 = inc1;

	if( hipMalloc(&flag_p,sizeof(*flag_p)) != hipSuccess ){
		NERROR1("cuda malloc error getting flag word");
	}

	/* use 2d allocator for better stride? */
	if( hipMalloc(&filled,len.d5_dim[1]*len.d5_dim[2]) != hipSuccess ){
		NERROR1("cuda malloc error getting filled array");
	}

	/* set filled to zero */
	//CLEAR_CUDA_ERROR2("h_sp_ifl","zeroit")
	CLEAR_CUDA_ERROR("zeroit")
	zeroit<<< NN_GPU >>>(szarr,filled,len);
	//CHECK_CUDA_ERROR("h_sp_ifl","zeroit")
	CHECK_CUDA_ERROR(h_sp_ifl: zeroit)

	// Get the value at the seed point
	f_p = (float *)OBJ_DATA_PTR(dp);
	f_p += x + y * inc1.d5_dim[2];

	e = hipMemcpy(&v, f_p, sizeof(v), hipMemcpyDeviceToHost);
	CHECK_CUDA_RETURN_VAL("hipMemcpy device to host");

	// Fill the seed point
	b_one = 1;
	e = hipMemcpy(filled+x+y*len.d5_dim[1], &b_one, sizeof(b_one),
						hipMemcpyHostToDevice);
	CHECK_CUDA_RETURN_VAL("hipMemcpy host to device");

	e = hipMemcpy(f_p, &fill_val, sizeof(fill_val),
						hipMemcpyHostToDevice);
	CHECK_CUDA_RETURN_VAL("hipMemcpy host to device");


	n_iterations=0;
	do {
		/* Clear the flag */
		h_flag = 0;
		e = hipMemcpy(flag_p, &h_flag, sizeof(h_flag),
						hipMemcpyHostToDevice);
		CHECK_CUDA_RETURN_VAL("hipMemcpy host to device");

		//CLEAR_CUDA_ERROR2("h_sp_ifl","g_sp_ifl_incs")
		CLEAR_CUDA_ERROR("g_sp_ifl_incs")
		g_sp_ifl_incs<<< NN_GPU >>>
		(szarr,(float *)OBJ_DATA_PTR(dp),inc1,filled,inc2,len,v,tol,fill_val,flag_p);
		//CHECK_CUDA_ERROR("h_sp_ifl","g_sp_ifl_incs")
		CHECK_CUDA_ERROR(h_sp_ifl: g_sp_ifl_incs)

		// download flag to see what happened.
		e = hipMemcpy(&h_flag, flag_p, 1,
						hipMemcpyDeviceToHost);
		CHECK_CUDA_RETURN_VAL("hipMemcpy device to host");
		n_iterations++;
	} while( h_flag );

	if( verbose ){
		sprintf(DEFAULT_ERROR_STRING,"Fill completed after %d iterations",n_iterations);
		NADVISE(DEFAULT_ERROR_STRING);
	}
}

void h_sp_ifl2( Data_Obj *dp, int seed_x, int seed_y, float tol, float fill_val )
{
	BLOCK_VARS_DECLS
	hipError_t e;
	dim5 len, inc1, inc2;
	unsigned char *filled, b_one;
	float *f_p, v;
	int n_iterations;
	Vector_Args va1, *vap=(&va1);
	dim5 szarr;

	len.d5_dim[1] = OBJ_COLS(dp);
	len.d5_dim[2] = OBJ_ROWS(dp);

	//GET_MAX_THREADS(dp)
	SETUP_BLOCKS_XYZ(OBJ_PFDEV(dp))

	inc1.d5_dim[1] = OBJ_TYPE_INC(dp,1);
	inc1.d5_dim[2] = OBJ_TYPE_INC(dp,2);
	inc1.d5_dim[0] = inc1.d5_dim[3] = inc1.d5_dim[4] = 0;
	inc2 = inc1;

	/* use 2d allocator for better stride? */
	if( hipMalloc(&filled,len.d5_dim[1]*len.d5_dim[2]) != hipSuccess ){
		NERROR1("cuda malloc error getting filled array");
	}

	/* set filled to zero */
	//CLEAR_CUDA_ERROR2("h_sp_ifl2","zeroit")
	CLEAR_CUDA_ERROR("zeroit")
	zeroit<<< NN_GPU >>>(szarr,filled,len);
	//CHECK_CUDA_ERROR("h_sp_ifl2","zeroit")
	CHECK_CUDA_ERROR(h_sp_ifl2: zeroit)

	// Get the value at the seed point
	f_p = (float *)OBJ_DATA_PTR(dp);
	f_p += seed_x + seed_y * inc1.d5_dim[2];

	e = hipMemcpy(&v, f_p, sizeof(v), hipMemcpyDeviceToHost);
	CHECK_CUDA_RETURN_VAL("hipMemcpy device to host");

	// Fill the seed point
	b_one = 1;
	e = hipMemcpy(filled+seed_x+seed_y*len.d5_dim[1], &b_one, sizeof(b_one),
						hipMemcpyHostToDevice);
	CHECK_CUDA_RETURN_VAL("hipMemcpy host to device");
	e = hipMemcpy(f_p, &fill_val, sizeof(fill_val),
						hipMemcpyHostToDevice);
	CHECK_CUDA_RETURN_VAL("hipMemcpy host to device");

	e = hipMemcpyToSymbol(HIP_SYMBOL(fill_value), &fill_val, sizeof(float));
	CHECK_CUDA_RETURN_VAL("hipMemcpyToSymbol");
	e = hipMemcpyToSymbol(HIP_SYMBOL(tolerance), &tol, sizeof(float));
	CHECK_CUDA_RETURN_VAL("hipMemcpyToSymbol");
	e = hipMemcpyToSymbol(HIP_SYMBOL(test_value), &v, sizeof(float));
	CHECK_CUDA_RETURN_VAL("hipMemcpyToSymbol");

	n_iterations=0;
	for( n_iterations = 0 ; n_iterations < 300 ; n_iterations++ ){

		//CLEAR_CUDA_ERROR2("h_sp_ifl2","g_sp_ifl2_incs")
		CLEAR_CUDA_ERROR("g_sp_ifl2_incs")
		g_sp_ifl2_incs<<< NN_GPU >>>
		(szarr,(float *)OBJ_DATA_PTR(dp),inc1,filled,inc2,len);
		//CHECK_CUDA_ERROR("h_sp_ifl2","g_sp_ifl2_incs")
		CHECK_CUDA_ERROR(h_sp_ifl2:  g_sp_ifl2_incs)

	}


	if( verbose ){
		sprintf(DEFAULT_ERROR_STRING,"Fill completed after %d iterations",n_iterations);
		NADVISE(DEFAULT_ERROR_STRING);
	}
}
#endif // FOOBAR


#endif /* HAVE_CUDA */

